#include <jrand.c>

// Function “ Random get_random_with_structure_seed” 
//Requires the potential world seed, an X value and a Z value and a generation chunk seed and
//salt 
static inline Random get_random_with_structure_seed (long world_seed, int x, int z, int salt) 

//Do not get confused by the big scary numbers 
//We are just multiplying the X and Z by a big scary number (addressed in SEED RNG) than   // the adding world seed and salt to this value. 
{ return get_random(x * 341873128712L + z * 132897987541L + world_seed + salt);}

// Minecraft fortress function 
static inline int fortress_at(long world_seed, int x, int z) {
 //Region size of 27 
 int rSize = 27;
 //separation distance of 4 
 int rSep = 4;
 int constrained_x = x < 0 ? x - rSize + 1 : x;
 int constrained_z = z < 0 ? z - rSize + 1 : z;
 int rx = constrained_x / rSize;
 int rz = constrained_z / rSize;
 Random r = get_random_with_structure_seed(world_seed, rx, rz, 30084232);
 if (random_next_int(&r, rSize - rSep) != x - rx * rSize) return 0;
 if (random_next_int(&r, rSize - rSep) != z - rz * rSize) return 0;
 if (random_next_int(&r,5) >= 2) return 0;
 return 1;
}
// kernel function 
__kernel void start(ulong offset, ulong stride, __global ulong *seeds, __global ushort *ret) {
 size_t id = get_global_id(0);
 uchar max_count = 0;
 uchar max_last = 0;
 ulong seed_base = (offset + id) * stride;
 for (ulong i = 0; i < stride; i++) {
  ulong worldSeed = seed_base|i;

   //fortress data if cPos cords continue 
  if (!fortress(worldSeed,-9, 20))continue; 
  if (!fortress(worldSeed,-10, 20))continue; 
  if (!fortress(worldSeed,-11, 20))continue; 
  if (!fortress (worldSeed,-12, 20))continue; 
  if (!fortress(worldSeed,-6, 19))continue; 
  if (!fortress(worldSeed,-6, 18))continue;
  if (!fortress(worldSeed,-6, 20))continue; 
  if (!fortress(worldSeed,-6, 21))continue; 
  if (!fortress(worldSeed,-8, 20))continue; 
  if (!fortress (worldSeed,-5, 39))continue; 
  if (!fortress(worldSeed,-5, 20))continue; 
  if (!fortress(worldSeed,-6, 22))continue; 
 
// 
 max_count++;
  seeds[id] = worldSeed;
 }
 ret[id] = (max_count << 8) | max_last;
}
